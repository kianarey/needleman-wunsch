#include "hip/hip_runtime.h"
#ifndef _NW_SINGLE_KERNEL_H_
#define _NW_SINGLE_KERNEL_H_

#define BLOCK_SIZE 32

#include "Needleman_Wunsch_CPU.h"

__global__ void nw_single_kernel(int* scoreMatrix, char* seq1, char* seq2, int rows, int cols, int startCol, int startRow) {

    // Calculate row index of scoreMatrix element
    int curRow = blockIdx.y * blockDim.y + threadIdx.y + startRow;

    // Calculate column index of scoreMatrix element
    int curCol = blockIdx.x * blockDim.x + threadIdx.x + startCol;

	int startDiag = startCol + startRow;
	int endDiag = startDiag + 2 * BLOCK_SIZE - 1;

    for (int curDiag = startDiag; curDiag < endDiag; curDiag++) {
        // sequentially calculate each element in diagonals
        __syncthreads();

        // verify element is part of diagonal and calculate score
        if (curRow + curCol == curDiag && curRow < rows && curCol < cols && curRow > 0 && curCol > 0 && curRow < startRow + BLOCK_SIZE && curCol < startCol + BLOCK_SIZE) {
			// calculate score from going down, this introduces a gap
			int	down = scoreMatrix[(curRow - 1) * cols + curCol] + GAP;

			// calculate score from going right, this introduces a gap
			int	right = scoreMatrix[curRow * cols + (curCol - 1)] + GAP;

			// calculate score from going diagonal
			int	diag = 0;
			if (seq1[curRow - 1] == seq2[curCol - 1]) {
				//seq1 row and seq2 col nucleotides match
				diag = scoreMatrix[(curRow - 1) * cols + (curCol - 1)] + MATCH;
			}
			else {
				//seq1 row and seq2 col nucleotides do not match
				diag = scoreMatrix[(curRow - 1) * cols + (curCol - 1)] + MISMATCH;
			}

			// update cell value scoreand trace values based on highest score
			if (diag >= down && diag >= right) {
				//diag produced best score
				scoreMatrix[curRow * cols + curCol] = diag;
			}
			else if (right >= diag && right >= down) {
				// right produced best score
				scoreMatrix[curRow * cols + curCol] = right;
			}
			else {
				// down produced best score
				scoreMatrix[curRow * cols + curCol] = down;
			}
        }
    }
}

__global__ void initialize_matrix(int* scoreMatrix, int rows, int cols) {

	int index = blockDim.x * blockIdx.x + threadIdx.x;

	// set gap alignment values for row 0
	if (index < cols) {
		scoreMatrix[index] = index * GAP;
	}

	__syncthreads();

	// set gap alignment values for col 0
	if (index < rows) {
		scoreMatrix[index * cols] = index * GAP;
	}

}

__global__ void nw_single_diag_kernel(int* scoreMatrix, char* seq1, char* seq2, int rows, int cols, int blockDiag, int blockCols) {

	// Value based on top left cell in block
	int startRow;
	int startCol;
	if (blockDiag <= blockCols) {
		startRow = blockIdx.y * blockDim.y;
		startCol = (blockDiag - 1 - blockIdx.y) * blockDim.x;
	}
	else {
		// requires additional offset
		startRow = (blockIdx.y + blockDiag - blockCols) * blockDim.y;
		startCol = (blockCols - 1 - blockIdx.y) * blockDim.x;
	}

	// Calculate row index of scoreMatrix element
	int curRow = threadIdx.y + startRow;

	// Calculate column index of scoreMatrix element
	int curCol = threadIdx.x + startCol;

	int startDiag = startCol + startRow;
	int endDiag = startDiag + blockDim.x + blockDim.y - 1;

	for (int curDiag = startDiag; curDiag < endDiag; curDiag++) {
		// sequentially calculate each element in diagonals
		__syncthreads();

		// verify element is part of diagonal and calculate score
		if (curRow + curCol == curDiag && curRow < rows && curCol < cols && curRow > 0 && curCol > 0 && curRow < startRow + blockDim.y && curCol < startCol + blockDim.y) {
			// calculate score from going down, this introduces a gap
			int	down = scoreMatrix[(curRow - 1) * cols + curCol] + GAP;

			// calculate score from going right, this introduces a gap
			int	right = scoreMatrix[curRow * cols + (curCol - 1)] + GAP;

			// calculate score from going diagonal
			int	diag = 0;
			if (seq1[curRow - 1] == seq2[curCol - 1]) {
				//seq1 row and seq2 col nucleotides match
				diag = scoreMatrix[(curRow - 1) * cols + (curCol - 1)] + MATCH;
			}
			else {
				//seq1 row and seq2 col nucleotides do not match
				diag = scoreMatrix[(curRow - 1) * cols + (curCol - 1)] + MISMATCH;
			}

			// update cell value scoreand trace values based on highest score
			if (diag >= down && diag >= right) {
				//diag produced best score
				scoreMatrix[curRow * cols + curCol] = diag;
			}
			else if (right >= diag && right >= down) {
				// right produced best score
				scoreMatrix[curRow * cols + curCol] = right;
			}
			else {
				// down produced best score
				scoreMatrix[curRow * cols + curCol] = down;
			}
		}
	}
}



__global__ void nw_single_diag_kernel_shared(int* scoreMatrix, char* seq1, char* seq2, int rows, int cols, int blockDiag, int blockCols) {

	// every thread in the block will place an element into seq1_shared and seq2_shared
	__shared__ char seq1_shared[BLOCK_SIZE*BLOCK_SIZE];
	__shared__ char seq2_shared[BLOCK_SIZE*BLOCK_SIZE];

	// Value based on top left cell in block
	int startRow;
	int startCol;
	if (blockDiag <= blockCols) {
		startRow = blockIdx.y * blockDim.y;
		startCol = (blockDiag - 1 - blockIdx.y) * blockDim.x;
	}
	else {
		// requires additional offset
		startRow = (blockIdx.y + blockDiag - blockCols) * blockDim.y;
		startCol = (blockCols - 1 - blockIdx.y) * blockDim.x;
	}

	// Calculate row index of scoreMatrix element
	int curRow = threadIdx.y + startRow;

	// Calculate column index of scoreMatrix element
	int curCol = threadIdx.x + startCol;

	int startDiag = startCol + startRow;
	int endDiag = startDiag + 2 * BLOCK_SIZE - 1;

	// Variables that flag whether the current block is within the first row/column of blocks
	// This becomes important when initializing the matrix
	int top_block_row;
	int left_block_col;

	// These are used to compare elements between the sequences in later kernel code
	char seq1_char;
	char seq2_char;

	if(curRow < rows){
		if(startRow != 0){
			seq1_shared[threadIdx.x + threadIdx.y*BLOCK_SIZE] = seq1[threadIdx.x + threadIdx.y*BLOCK_SIZE + startRow - 1];
			top_block_row = 0;
		}
		else{
			seq1_shared[threadIdx.x + threadIdx.y*BLOCK_SIZE] = seq1[threadIdx.x + threadIdx.y*BLOCK_SIZE + startRow];
			top_block_row = 1;
		}
	}
	if(curCol < cols){
		if(startCol != 0){
			seq2_shared[threadIdx.x + threadIdx.y*BLOCK_SIZE] = seq2[threadIdx.x + threadIdx.y*BLOCK_SIZE + startCol - 1];
			left_block_col = 0;
		}
		else{
			seq2_shared[threadIdx.x + threadIdx.y*BLOCK_SIZE] = seq2[threadIdx.x + threadIdx.y*BLOCK_SIZE + startCol];
			left_block_col = 1;
		}
	}

	__syncthreads();


	for (int curDiag = startDiag; curDiag < endDiag; curDiag++) {
		// sequentially calculate each element in diagonals
		__syncthreads();

		// verify element is part of diagonal and calculate score
		if (curRow + curCol == curDiag && curRow < rows && curCol < cols && curRow > 0 && curCol > 0 && curRow < startRow + BLOCK_SIZE && curCol < startCol + BLOCK_SIZE) {
			// calculate score from going down, this introduces a gap
			int	down = scoreMatrix[(curRow - 1) * cols + curCol] + GAP;

			// calculate score from going right, this introduces a gap
			int	right = scoreMatrix[curRow * cols + (curCol - 1)] + GAP;

			// calculate score from going diagonal
			int	diag = 0;

			// if it's not a block on the first row
			if(top_block_row == 0)
				seq1_char = seq1_shared[threadIdx.y];
			else
				seq1_char = seq1_shared[threadIdx.y - 1];

			if(left_block_col == 0)
				seq2_char = seq2_shared[threadIdx.x];
			else
				seq2_char = seq2_shared[threadIdx.x - 1];

			if (seq1_char == seq2_char) {
				//seq1 row and seq2 col nucleotides match
				diag = scoreMatrix[(curRow - 1) * cols + (curCol - 1)] + MATCH;
			}
			else {
				//seq1 row and seq2 col nucleotides do not match
				diag = scoreMatrix[(curRow - 1) * cols + (curCol - 1)] + MISMATCH;
			}

			// update cell value scoreand trace values based on highest score
			if (diag >= down && diag >= right) {
				//diag produced best score
				scoreMatrix[curRow * cols + curCol] = diag;
			}
			else if (right >= diag && right >= down) {
				// right produced best score
				scoreMatrix[curRow * cols + curCol] = right;
			}
			else {
				// down produced best score
				scoreMatrix[curRow * cols + curCol] = down;
			}
		}
	}
}

void launch_single_block_kernel(int* scoreMatrix, char* seq1, char* seq2, int rows, int cols) {
	// fill in first row & col of the matrix
	initialize_matrix << < ceil(max(cols, rows) / (float)256), 256 >> > (scoreMatrix, rows, cols);

    // requires 1 thread/element
	// 1 kernel launch per block (sequentially calculates blocks)
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
	for (int startRow = 1; startRow < rows; startRow += BLOCK_SIZE) {
		for (int startCol = 1; startCol < cols; startCol += BLOCK_SIZE) {
			nw_single_kernel << < 1, dimBlock >> > (scoreMatrix, seq1, seq2, rows, cols, startCol, startRow);
		}
	}
}

void launch_diag_block_kernel(int* scoreMatrix, char* seq1, char* seq2, int rows, int cols) {
	// fill in first row & col of the matrix
	initialize_matrix << < ceil(max(cols, rows) / (float)256), 256 >> > (scoreMatrix, rows, cols);

	// requires 1 thread/element
	// 1 kernel launch per diagonal of blocks (sequentially calculates diagonal)
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
	int blockRows = ceil((float)rows / BLOCK_SIZE);
	int blockCols = ceil((float)cols / BLOCK_SIZE);
	for (int diag = 1; diag < blockRows + blockCols ; diag++) {
		// determine number of blocks along diagonal
		int diagBlocks;
		if (blockRows <= blockCols) {
			if (diag <= blockRows) {
				diagBlocks = diag;
			}
			else if (diag <= blockCols) {
				diagBlocks = blockRows;
			}
			else {
				diagBlocks = blockRows - (diag - blockCols);
			}
		}
		else {
			if (diag <= blockCols) {
				diagBlocks = diag;
			}
			else if (diag <= blockCols) {
				diagBlocks = blockCols;
			}
			else {
				diagBlocks = blockCols - (diag - blockRows);
			}
		}
		dim3 dimGrid(1, diagBlocks, 1);
		nw_single_diag_kernel_shared << < dimGrid, dimBlock >> > (scoreMatrix, seq1, seq2, rows, cols, diag, blockCols);
	}
}

/*
======================================
	Multi Kernel Implementation / Start
======================================
*/


// Utility function to compute a given cell and write result to scoreMatrix[cur_row *cols + cur_col]
__device__ void compute_cell(int* scoreMatrix, char* seq1, char* seq2, int rows, int cols, int cur_row, int cur_col ) {


	// // calculate score from going down, this introduces a gap
	int	down = scoreMatrix[(cur_row - 1) * cols + cur_col] + GAP;

	// // calculate score from going right, this introduces a gap
	int	right = scoreMatrix[(cur_row) * cols + cur_col - 1] + GAP;

	// calculate score from going diagonal
	int	diag = 0;
	if (seq1[cur_row-1] == seq2[cur_col-1]) {
		//seq1 row and seq2 col nucleotides match
		diag = scoreMatrix[(cur_row -1) * cols + (cur_col - 1)] + MATCH;
	}
	else {
		//seq1 row and seq2 col nucleotides do not match
		diag = scoreMatrix[(cur_row-1)* cols + cur_col-1] + MISMATCH;
	}

	// update cell value scoreand trace values based on highest score
	if (diag >= down && diag >= right) {
		//diag produced best score
		scoreMatrix[cur_row *cols + cur_col] = diag;
	}
	else if (right >= diag && right >= down) {
		// right produced best score
		scoreMatrix[cur_row *cols + cur_col] = right;
	}
	else {
		// down produced best score
		scoreMatrix[cur_row *cols + cur_col] = down;
	}
}

/*
	Launch a single block of 32 threads. 
	Iterates through first & last 32 diagonals of the matrix 
	Single warp, no __syncthreads needed.
*/
__global__ void warp_level_diag(int* scoreMatrix, char* seq1, char* seq2, int rows, int cols, bool first_half) {
	int cur_row, cur_col;
	int diagIndex;
	int maxNumDiagonals = rows; // Assuming a square matrix . TODO: make this work for rectangular matrices too
	for (int i =0; i < 32; i++) {
		// For the first 32 diagonals of the matrix
		// Calculate the current row & col for each iteration
		if (first_half) {
			diagIndex = i;
			cur_row = threadIdx.x+1;
			cur_col =  diagIndex - threadIdx.x;
		} 
		// For the last 32 diagonals of the matrix
		// Calculate the current row & col for each iteration
		else {
			diagIndex = 32 - i; 
			cur_row  = maxNumDiagonals - diagIndex + threadIdx.x;
			cur_col = cols -1  - threadIdx.x;
		}
		// If thread maps to one of the diagonal cells, compute the value 
		if(threadIdx.x < diagIndex) {
			compute_cell(scoreMatrix, seq1, seq2, rows, cols, cur_row, cur_col);
		}
	}
}

/*
	Iterates through first & last 1024 diagonals of the matrix 
	If the matrix width/lenth is smaller than 1024 then iterate 
	through the entire matrix.
	Threads need to be synched at each iteration of the diagonal.
*/
__global__ void block_level_diag(int* scoreMatrix, char* seq1, char* seq2, int rows, int cols, bool first_half) {
	int cur_row, cur_col;
	
	int index = blockIdx.x *  blockDim.x * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x;

	/*
		For the first 1024 diagonals of the matrix
		Calculate the current row & col for each iteration
		If the matrix width/lenth is smaller than 1024
		then iterate through the entire upper diagonal of the matrix
			Assuming a square matrix. TODO: make this work for rectangular matrices too
	*/
	int maxNumDiagonals = min(1024, rows-1);  
	 if (first_half) {
		for (int i =32; i < maxNumDiagonals; i++) {
			cur_row = index + 1;
			cur_col = i - index;		
			// If thread maps to one of the diagonal cells, compute the value 
			if(index < i){
				compute_cell(scoreMatrix, seq1, seq2, rows, cols, cur_row, cur_col);
			}
			__syncthreads();
		}
	} 
	
	/*
		For the last 1024 diagonals of the matrix,
		Calculate the current row & col for each iteration
		If the matrix width/lenth is smaller than 1024
		then iterate through the entire lower diagonal of the matrix
	*/
	else {
		for (int i =maxNumDiagonals-1; i >= 31; i--) {
			cur_row = (rows-1) - i + index ;
			cur_col = (rows-1) - index ;
			// If thread maps to one of the diagonal cells, compute the value 
			if(index <= i ) {
				compute_cell(scoreMatrix, seq1, seq2, rows, cols, cur_row, cur_col);			
			}
			__syncthreads();
		}
	}
}

/*
	Runs through a single digonal with multiple blocks, assigning each thread to compute a single cell.
*/
__global__ void multi_block_level_diag(int* scoreMatrix, char* seq1, char* seq2, int rows, int cols, int diagIndex, bool first_half) {
	int cur_row, cur_col;
	int index = blockIdx.x *  blockDim.x * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x;

	// The rest of the upper diagonal of the matrix
	if (first_half) {
		cur_row = index + 1;
		cur_col = diagIndex - index;

		// If thread maps to one of the diagonal cells, compute the value 
		if(index < diagIndex){
			compute_cell(scoreMatrix, seq1, seq2, rows, cols, cur_row, cur_col);
		}
	} 
	// The rest of the lower diagonal of the matrix. 
	else {
		cur_row = rows - 1 - diagIndex + index ;
		cur_col = rows - 1 - index ;

		// If thread maps to one of the diagonal cells, compute the value 
		if(index <= diagIndex ) {
			compute_cell(scoreMatrix, seq1, seq2, rows, cols, cur_row, cur_col);			
		}	
	}
}

void launch_multi_kernel_diag(int* scoreMatrix, char* seq1, char* seq2, int rows, int cols) {
	initialize_matrix << < ceil(max(cols, rows) / (float)256), 256 >> > (scoreMatrix, rows, cols);
	dim3 dimBlock(32, 32, 1);
	dim3 dimGrid(ceil(rows / (double)1024), 1, 1);

	// Single Kernel launch to go trhough first 32 diagonals of the matrix
	warp_level_diag << < 1, 32 >> > (scoreMatrix, seq1, seq2, rows, cols, true);
	
	
	// Single Kernel launch to go trhough first 1024 diagonals of the matrix
	block_level_diag << < dimGrid, dimBlock >> > (scoreMatrix, seq1, seq2, rows, cols, true);

	// Rest of the diagonals in the upper corner of the matrix
	// Launches multiples block of 32x32 threads. Each Kernel call computes a diagonal.
	for(int i =1024; i < rows; i++) {
		multi_block_level_diag << < dimGrid, dimBlock >> > (scoreMatrix, seq1, seq2, rows, cols, i, true);
	}

	// Rest of the diagonals in the lower corner of the matrix
	// Launches multiples block of 32x32 threads. Each Kernel call computes a diagonal.
	for(int i = rows-2; i >= 1024; i--) {
		multi_block_level_diag << < dimGrid, dimBlock >> > (scoreMatrix, seq1, seq2, rows, cols, i, false);

	}
	
	// Single Kernel launch to go trhough last 1024 diagonals of the matrix
	block_level_diag << < dimGrid, dimBlock >> > (scoreMatrix, seq1, seq2, rows, cols, false);
	
	// Single Kernel launch to go trhough last 32 diagonals of the matrix
	warp_level_diag << < 1, 32 >> > (scoreMatrix, seq1, seq2, rows, cols, false);
}


/*
======================================
	Multi Kernel Implementation / End
======================================
*/


void computeOnDevice(int* scoreMatrix, char* seq1, char* seq2, int rows, int cols, int n, int kernel_mode) {

	// setup code
	int* d_scoreMatrix;
	char* d_seq1, * d_seq2;

	hipMalloc((void**)&d_scoreMatrix, sizeof(int) * (rows * cols));
	hipMalloc((void**)&d_seq1, sizeof(char) * (rows - 1));
	hipMalloc((void**)&d_seq2, sizeof(char) * (cols - 1));
	hipMemcpy(d_seq1, seq1, sizeof(char) * (rows - 1), hipMemcpyHostToDevice);
	hipMemcpy(d_seq2, seq2, sizeof(char) * (cols - 1), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	printf("\tTiming 'Needleman_Wunsch_GPU' started\n");

	
	if(kernel_mode == 1 || kernel_mode == 2){
		// Launch kernel call for GPU calculation
		hipEventRecord(start);
		for (int i = 0; i < n; i++) {
			if(kernel_mode==1){
				launch_diag_block_kernel(d_scoreMatrix, d_seq1, d_seq2, rows, cols);
			}
			else{
				launch_multi_kernel_diag(d_scoreMatrix, d_seq1, d_seq2, rows, cols);
			}
		}

		hipEventRecord(stop);
		hipEventSynchronize(stop);
	}

	else if(kernel_mode == 3){
		hipEventRecord(start);

		launch_single_block_kernel(d_scoreMatrix, d_seq1, d_seq2, rows, cols);

		hipEventRecord(stop);
		hipEventSynchronize(stop);
	}

	printf("\tTiming 'Needleman_Wunsch_GPU' ended\n");

	float device_ms = 0;
	hipEventElapsedTime(&device_ms, start, stop);

	printf("\t%i iterations = %f\n", n, device_ms / 1000);

	// Teardown code
	hipMemcpy(scoreMatrix, d_scoreMatrix, sizeof(int) * (rows * cols), hipMemcpyDeviceToHost);
	hipFree(d_scoreMatrix);
	hipFree(d_seq1);
	hipFree(d_seq2);
}

#endif // #ifndef _NW_SINGLE_KERNEL_H_
